#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <chrono>

// matrix dimensions
const int heightA = 4;
const int widthA = 3;
const int heightB = 3;
const int widthB = 2;
const int arraySizeA = heightA * widthA;
const int arraySizeB = heightB * widthB;
const int arraySizeC = heightA * widthB;

// each thread computes one element of C
__global__ void matrixMulKernel(int* c, const int* a, const int* b)
{
    // thread index within the block
    int row = threadIdx.y;
    int col = threadIdx.x;

    // check if this thread should compute a result element
    if (row < heightA && col < widthB) {
        int sum = 0;
        // dot product loop
        for (int k = 0; k < widthA; k++) {
            sum += a[row * widthA + k] * b[k * widthB + col];
        }
        c[row * widthB + col] = sum;
    }
}

// helper function for using cuda to perform matrix multiplication
hipError_t matrixMulWithCuda(int* c, const int* a, const int* b);

int main()
{
    // define input matrices
    const int a[arraySizeA] = {
        1, 2, 3,
        4, 5, 6,
        7, 8, 9,
        10, 11, 12
    };

    const int b[arraySizeB] = {
        1, 2,
        3, 4,
        5, 6
    };

    int c[arraySizeC] = { 0 };

    auto start = std::chrono::high_resolution_clock::now();
    hipError_t cudaStatus = matrixMulWithCuda(c, a, b);
    auto end = std::chrono::high_resolution_clock::now();

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "matrixMulWithCuda failed!");
        return 1;
    }

    // fancy way to print the matrices
    std::cout << "Matrix A (" << heightA << "x" << widthA << "):" << std::endl;
    for (int i = 0; i < heightA; i++) {
        for (int j = 0; j < widthA; j++) {
            std::cout << a[i * widthA + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << "\nMatrix B (" << heightB << "x" << widthB << "):" << std::endl;
    for (int i = 0; i < heightB; i++) {
        for (int j = 0; j < widthB; j++) {
            std::cout << b[i * widthB + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << "\nResult Matrix C = A * B (" << heightA << "x" << widthB << "):" << std::endl;
    for (int i = 0; i < heightA; i++) {
        for (int j = 0; j < widthB; j++) {
            std::cout << c[i * widthB + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << "\nCUDA Time taken: " << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() << " microseconds" << std::endl;

    // reset cuda device
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t matrixMulWithCuda(int* c, const int* a, const int* b)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;


    cudaStatus = hipMalloc((void**)&dev_c, arraySizeC * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, arraySizeA * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, arraySizeB * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_a, a, arraySizeA * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, arraySizeB * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    dim3 threadsPerBlock(widthB, heightA);
    matrixMulKernel << <1, threadsPerBlock >> > (dev_c, dev_a, dev_b);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching matrixMulKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(c, dev_c, arraySizeC * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

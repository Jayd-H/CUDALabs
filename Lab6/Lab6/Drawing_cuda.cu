#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_vector_types.h>

 // includes, cuda
#include <hip/hip_runtime_api.h>

typedef unsigned int uint;
typedef unsigned char uchar;


hipArray* d_imageArray = 0;


__global__ void d_render(uchar4* d_output, uint width, uint height) {
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint i = __umul24(y, width) + x;
    if ((x < width) && (y < height)) {

        // normalise to 1, -1
        float u = x / (float)width;
        float v = y / (float)height;
        u = 2.0f * u - 1.0f;
        v = -(2.0f * v - 1.0f);

        // scale coords to 2, -2 to zoom in rather than 4, -4
        u *= 2.0f;
        v *= 2.0f;

        // julia set
        float2 z = make_float2(u, v);

        // For Julia set, c is fixed rather than being the starting point
        float2 c = make_float2(0.25f, 0.5f);

        float r = 0.0f;
        float color = 1.0f;

        // iterate 30 times (given by the lab pseudocode)
        for (int j = 0; j < 30; j++) {
            // z = z^2 + c
            float2 z_squared = make_float2(z.x * z.x - z.y * z.y, 2.0f * z.x * z.y);
            z = make_float2(z_squared.x + c.x, z_squared.y + c.y);

            // calculate magnitude of z
            r = sqrtf(z.x * z.x + z.y * z.y);

            // if magnitude of z is greater than 5, the pixel is not in the set
            if (r > 5.0f) {
                color = 0.0f;
                break;
            }
        }
        // set pixel colour based on if its in the set or not
        if (color > 0.0f) {
            // pixels in the set are red
            d_output[i] = make_uchar4(0, 0, 0xff, 0);
        }
        else {
            // pixels that are not are in black
            d_output[i] = make_uchar4(0, 0, 0, 0);
        }
    }
}


extern "C" void freeTexture() {

    checkCudaErrors(hipFreeArray(d_imageArray));
}

// render image using CUDA
extern "C" void render(int width, int height,  dim3 blockSize, dim3 gridSize,
     uchar4 * output) {


            d_render << <gridSize, blockSize >> > (output, width, height);


    getLastCudaError("kernel failed");
}

#endif